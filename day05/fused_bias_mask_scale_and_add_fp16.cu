#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <algorithm>
typedef __half half;
typedef __half2 half2;

template<typename T>
struct MaskScaleAndElementwiseAddFunctor {
    MaskScaleAndElementwiseAddFunctor(const uint8_t* mask, const T* add_val, float scale)
        : mask(mask), add_val(add_val), scale(scale) {}

    __device__ T Compute(T x, int64_t i) const {
            return x * static_cast<T>(static_cast<bool>(mask[i]) * scale) + add_val[i];
    }
    const uint8_t* mask;
    const T* add_val;
    float scale;
};

template<>
struct MaskScaleAndElementwiseAddFunctor<half> {
    MaskScaleAndElementwiseAddFunctor(const uint8_t* mask, const half* add_val, float scale)
      : mask(mask), add_val(add_val), scale(scale) {}

    __device__ half2 ComputeHalf2(half2 x, int64_t i) const {
        const char2* mask_c2 = reinterpret_cast<const char2*>(mask);
        const half2* add_val_h2 = reinterpret_cast<const half2*>(add_val);
        char2 mask_val = mask_c2[i]; // * Vectorized read
        half2 one_or_zero_h2; // * Vectorized read
        half2 h2_scale = __float2half2_rn(scale); // * float->half2, e.g. 1.0 => (1.0, 1.0)
        reinterpret_cast<half&>(one_or_zero_h2.x) = mask_val.x;
        reinterpret_cast<half&>(one_or_zero_h2.y) = mask_val.y;
        return __hadd2(__hmul2(__hmul2(x, one_or_zero_h2), h2_scale), add_val_h2[i]);
    }

    const uint8_t* mask;
    const half* add_val;
    float scale;
};

// biasAdd takes two inputs: x.shape={rows, cols}, bias.shape={cols}, so we need to use division and modulo in L59 to cycle through these cols bias values
template<typename FUNCTOR>
__global__ void FusedBiasAddCUDAKernelHalf2(FUNCTOR functor, const int elem_cnt,
                                        const int bias_size, const half* x, const half* bias,
                                        half* y) {
    const int h2_elem_cnt = elem_cnt / 2; // * Element count is halved since we're reading half2 instead of half
    const int h2_bias_size = bias_size / 2;
    const auto* x_h2 = reinterpret_cast<const half2*>(x); // * Cast to vector pointer
    const auto* bias_h2 = reinterpret_cast<const half2*>(bias);
    auto* y_h2 = reinterpret_cast<half2*>(y);  // Add type conversion for output pointer
    // * Ensure limited number of threads process all data
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < h2_elem_cnt;
        i += blockDim.x * gridDim.x) {
            half2 x_i = __hadd2(x_h2[i], bias_h2[i % h2_bias_size]);
            y_h2[i] = functor.ComputeHalf2(x_i, i);  // y_h2 is now defined
    }
}

int main() {
    const int ele_cnt = 1000000;
    float scale = 0.5;
    uint8_t* mask_tensor = new uint8_t[ele_cnt];
    __half* add_val = new __half[ele_cnt];
    for (int i = 0; i < ele_cnt; i++) {
        mask_tensor[i] = (uint8_t)(i);
    }
    int bias_size = 10;

    __half *x = (__half*) malloc(sizeof(__half) * ele_cnt);
    __half *y = (__half*) malloc(sizeof(__half) * ele_cnt);
    __half *bias = (__half*) malloc(sizeof(__half) * bias_size);
    for (int i = 0; i < ele_cnt; i++)
    {
      x[i] = (__half)(i);
    }
    __half *d_x, *d_y, *d_bias;
    hipMalloc((void **)&d_x, ele_cnt * sizeof(__half));
    hipMalloc((void **)&d_y, ele_cnt * sizeof(__half));
    hipMalloc((void **)&d_bias, bias_size * sizeof(__half));
    hipMemcpy(d_x, x, sizeof(__half) * ele_cnt, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sizeof(__half) * ele_cnt, hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, sizeof(__half) * bias_size, hipMemcpyHostToDevice);

    uint8_t *d_mask_tensor;
    __half *d_add_val;
    hipMalloc((void **)&d_mask_tensor, ele_cnt * sizeof(uint8_t));
    hipMalloc((void **)&d_add_val, ele_cnt * sizeof(__half));
    hipMemcpy(d_mask_tensor, mask_tensor, sizeof(uint8_t) * ele_cnt, hipMemcpyHostToDevice);
    hipMemcpy(d_add_val, add_val, sizeof(__half) * ele_cnt, hipMemcpyHostToDevice);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int maxblocks = deviceProp.maxGridSize[0];
    int blockSize = 256;
    int gridSize = std::min((ele_cnt + blockSize - 1) / blockSize, maxblocks);
    MaskScaleAndElementwiseAddFunctor<half> mask_scale_elementwise_add_func(d_mask_tensor, d_add_val, scale);
    FusedBiasAddCUDAKernelHalf2<<<gridSize, blockSize>>>( mask_scale_elementwise_add_func, ele_cnt, bias_size, d_x, d_bias, d_y);

    hipMemcpy(y, d_y, sizeof(__half) * ele_cnt, hipMemcpyDeviceToHost);
    
    free(x);
    free(y);
    free(bias);
    delete add_val;
    add_val = nullptr;
    delete mask_tensor;
    mask_tensor = nullptr;
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_bias);
    hipFree(d_mask_tensor);
    hipFree(d_add_val);
}