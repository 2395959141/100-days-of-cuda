#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include "transpose.cu"

// 测试函数
void test_bandwidth(int M, int N) {
    // 计算矩阵大小
    size_t size = M * N * sizeof(float);
    
    // 分配主机内存
    float* h_in = (float*)malloc(size);
    float* h_out_v1 = (float*)malloc(size);
    float* h_out_v2 = (float*)malloc(size);
    float* h_out_v3 = (float*)malloc(size);
    
    // 初始化输入矩阵
    for (int i = 0; i < M * N; i++) {
        h_in[i] = (float)i;
    }
    
    // 分配设备内存
    float *d_in, *d_out_v1, *d_out_v2, *d_out_v3;
    hipMalloc(&d_in, size);
    hipMalloc(&d_out_v1, size);
    hipMalloc(&d_out_v2, size);
    hipMalloc(&d_out_v3, size);
    
    // 拷贝数据到设备
    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);
    
    // 设置线程块和网格大小
    dim3 blockDim(32, 32);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);
    
    // 创建CUDA事件用于计时
    hipEvent_t start, stop;
    
    // 测试v1版本
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < 100; i++) {
        matrix_shared_trans_v1<<<gridDim, blockDim>>>(d_in, d_out_v1, M, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds_v1 = 0;
    hipEventElapsedTime(&milliseconds_v1, start, stop);
    
    // 测试v2版本
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < 100; i++) {
        matrix_shared_trans_v2<<<gridDim, blockDim>>>(d_in, d_out_v2, M, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds_v2 = 0;
    hipEventElapsedTime(&milliseconds_v2, start, stop);
    
    // 测试v3版本
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < 100; i++) {
        matrix_shared_trans_v3<<<gridDim, blockDim>>>(d_in, d_out_v3, M, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds_v3 = 0;
    hipEventElapsedTime(&milliseconds_v3, start, stop);
    
    // 计算带宽
    double total_bytes = 2.0 * M * N * sizeof(float) * 100;
    double bandwidth_v1 = (total_bytes / (milliseconds_v1 / 1000.0)) / 1e9;
    double bandwidth_v2 = (total_bytes / (milliseconds_v2 / 1000.0)) / 1e9;
    double bandwidth_v3 = (total_bytes / (milliseconds_v3 / 1000.0)) / 1e9;
    
    // 输出结果
    std::cout << "矩阵大小: " << M << "x" << N << std::endl;
    std::cout << "v1带宽: " << bandwidth_v1 << " GB/s" << std::endl;
    std::cout << "v2带宽: " << bandwidth_v2 << " GB/s" << std::endl;
    std::cout << "v3带宽: " << bandwidth_v3 << " GB/s" << std::endl;
    std::cout << "------------------------" << std::endl;
    
    // 释放资源
    hipFree(d_in);
    hipFree(d_out_v1);
    hipFree(d_out_v2);
    hipFree(d_out_v3);
    free(h_in);
    free(h_out_v1);
    free(h_out_v2);
    free(h_out_v3);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    // 测试不同大小的矩阵
    test_bandwidth(1024, 1024);
    test_bandwidth(2048, 2048);
    test_bandwidth(4096, 4096);
    test_bandwidth(8192, 8192);
    test_bandwidth(16384, 16384);
    return 0;
}
